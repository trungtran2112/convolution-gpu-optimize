#include "hip/hip_runtime.h"
#include "cuda_manager.h"
#include<iostream>
using namespace std;

#define TILE_WIDTH 16

// Convolution forward kernel: Naive implementation
__global__ void conv_forward_kernel(const float *in, float *out, const float *weight,
                                    const int channel_in, const int channel_out,
                                    const int height_in, const int width_in, const int kernel_width)
{
    const int height_out = height_in - kernel_width + 1; //24
    const int width_out = width_in - kernel_width + 1; //24

    int height_grid = (height_out - 1) / TILE_WIDTH + 1; //2
    int width_grid = (width_out - 1) / TILE_WIDTH + 1; //2

    int sample_idx = blockIdx.z; //cho biết ảnh thứ mấy trong batch
    int feature_map_idx = blockIdx.x; //cho biết đang xét kernel thứ mấy

    int row = (blockIdx.y / width_grid) * TILE_WIDTH + threadIdx.y; //tính cái dòng hiện tại trong input
    int col = (blockIdx.y % width_grid) * TILE_WIDTH + threadIdx.x; //tính cái cột hiện tại trong input

    float sum = 0;

    if (row < height_out && col < width_out)
    {
      int hw_in = height_in * width_in; //28x28
      int hw_out = height_out * width_out; //24x24

      for (int i = 0; i < channel_in; i++)
      {
          for (int j = 0; j < kernel_width; j++)
          {
              for (int k = 0; k < kernel_width; k++)
              {
                  int pixel_row = row + j;
                  int pixel_col = col + k;
                  sum += in[sample_idx * channel_in * hw_in + i * hw_in + //sample_idx * channel_in * hw_in tính từ vị trí đầu đến channel khác
                              pixel_row * width_in + pixel_col] *           //i * hw_in chọn lớp ảnh
                          weight[feature_map_idx * channel_in * kernel_width * kernel_width +
                              i * kernel_width * kernel_width + j * kernel_width + k];
              }
          }
      }
      out[sample_idx * channel_out * hw_out + feature_map_idx * hw_out + row * width_out + col] = sum;
    }

    
}

__global__ void conv_forward_kernel_2(const float *X, float *out, const float *W,
                                      const int C_in, const int C_out,
                                      const int H_in, const int W_in, const int K)
{
  int m, h_base, w_base, h,w; 
  int X_tile_width = TILE_WIDTH + K-1; 
  extern __shared__ float shmem[]; 
  float* X_shared = &shmem[0]; 
  float* W_shared = &shmem[X_tile_width * X_tile_width];

  const int H_out = H_in - K + 1; //24
  const int W_out = W_in - K + 1; //24
  int W_grid = (W_out - 1) / TILE_WIDTH + 1; //2

  m = blockIdx.x; 
  h_base = (blockIdx.y / W_grid) * TILE_WIDTH; // vertical base out data index for the block 
  w_base = (blockIdx.y % W_grid) * TILE_WIDTH; // horizontal base out data index for the block  
  
  int tx = threadIdx.x; 
  int ty = threadIdx.y; 
  h = h_base + tx; 
  w = w_base + ty; 
  int sample_idx = blockIdx.z;
  float acc = 0.; 
  for (int c = 0; c < C_in; c++)
  {
    //load W vào shared memory
    if (( ty < K) && ( tx < K)) 
    {
      W_shared[ty * K + tx]= W[m * C_in * K * K + c * K * K + ty * K + tx];
      // load tile from X[n, c,…] into shared memory 
    }
    else
    {
      W_shared[ty * K + tx] = 0;
    }
    __syncthreads(); 

 
 //load từng block từ X sang shared_memory
    for (int i = h; i < h_base + X_tile_width; i += TILE_WIDTH) 
    { 
      for (int j = w; j < w_base + X_tile_width; j += TILE_WIDTH) 
      {
        if(i < H_in && j < W_in)
        {
          X_shared[(i - h_base)*X_tile_width + (j - w_base)] = X[sample_idx * C_in * H_in * W_in + W_in * H_in * c + i * W_in + j]; 
        }
        else
        {
          X_shared[(i - h_base)*X_tile_width + (j - w_base)] = 0;
        }
      }
    } 
    __syncthreads(); 

    //Tính tích chập trên 1 kernel
    for (int p=0; p<K; p++) 
    {
        for (int q=0; q<K; q++) {
          if (((ty+p)<X_tile_width) && ((tx+q)<X_tile_width)) {
            acc += X_shared[(ty+p)*X_tile_width+(tx+q)]*W_shared[p*K+q];
          }
        }
      }
    __syncthreads(); 
  
    if(m<C_out && h<H_out && w<W_out) 
    {
      out[sample_idx * C_out * H_out * W_out + m  * H_out * W_out + h * W_out + w] = acc;
    }
  }
}



__host__ void cuda_manager::conv_forward(const float *in, float *out, const float *weight,
                                         const int n_samples, const int channel_in, const int channel_out,
                                         const int height_in, const int width_in, const int kernel_width, const int n_streams,const int kernel)
{
    int height_out = height_in - kernel_width + 1;
    int width_out = width_in - kernel_width + 1;
    int size_in = n_samples * channel_in * height_in * width_in;
    int size_out = n_samples * channel_out * height_out * width_out;
    int size_weight = channel_out * channel_in * kernel_width * kernel_width;

    float *d_in;
    float *d_out;
    float *d_weight;
    CHECK(hipMalloc((void **)&d_in, size_in * sizeof(float)));
    CHECK(hipMalloc((void **)&d_out, size_out * sizeof(float)));
    CHECK(hipMalloc((void **)&d_weight, size_weight * sizeof(float)));
    CHECK(hipMemcpy(d_in, in, size_in * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_weight, weight, size_weight * sizeof(float), hipMemcpyHostToDevice));

    
    // Create "nStreams" device streams
    hipStream_t *streams = (hipStream_t *)malloc(n_streams * sizeof(hipStream_t));
    for (int i = 0; i < n_streams; i++)
        CHECK(hipStreamCreate(&(streams[i])));
    
    int n_samples_per_stream = (n_samples + n_streams - 1) / n_streams;
    int size_in_per_stream = n_samples_per_stream * channel_in * height_in * width_in;
    int size_out_per_stream = n_samples_per_stream * channel_out * height_out * width_out;
    int size_weight_per_stream = channel_out * channel_in * kernel_width * kernel_width;
    int size_in_per_sample = channel_in * height_in * width_in;
    int size_out_per_sample = channel_out * height_out * width_out;
    int size_weight_per_sample = channel_out * channel_in * kernel_width * kernel_width;

    // Set grid and block dimensions and launch the kernel
    int height_grid = (height_out - 1) / TILE_WIDTH + 1;
    int width_grid = (width_out - 1) / TILE_WIDTH + 1;
    int z = height_grid * width_grid;

    for (int i = 0; i < n_streams; i++)
    {
        int offset = i * n_samples_per_stream;
        int n_samples_per_stream_ = min(n_samples - offset, n_samples_per_stream);
        int size_in_per_stream = n_samples_per_stream_ * channel_in * height_in * width_in;
        int size_out_per_stream = n_samples_per_stream_ * channel_out * height_out * width_out;
        int size_in_per_sample = channel_in * height_in * width_in;
        int size_out_per_sample = channel_out * height_out * width_out;

        CHECK(hipMemcpyAsync(d_in + offset * size_in_per_sample, in + offset * size_in_per_sample, size_in_per_stream * sizeof(float), hipMemcpyHostToDevice, streams[i]));
        CHECK(hipMemcpyAsync(d_weight, weight, size_weight * sizeof(float), hipMemcpyHostToDevice, streams[i]));

        dim3 dimGrid(channel_out, z, n_samples_per_stream_);
        dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
        if (kernel == 1 || kernel == 2)
          conv_forward_kernel<<<dimGrid, dimBlock, 0, streams[i]>>>(d_in + offset * size_in_per_sample, d_out + offset * size_out_per_sample, d_weight, channel_in, channel_out, height_in, width_in, kernel_width);
        else if (kernel == 3)
        {
          int s_mem = (TILE_WIDTH+kernel_width-1) * (TILE_WIDTH+kernel_width-1)  * sizeof(float);
          conv_forward_kernel_2<<<dimGrid, dimBlock, s_mem , streams[i]>>>(d_in + offset * size_in_per_sample, d_out + offset * size_out_per_sample, d_weight, channel_in, channel_out, height_in, width_in, kernel_width);
        }
        CHECK(hipMemcpyAsync(out + offset * size_out_per_sample, d_out + offset * size_out_per_sample, size_out_per_stream * sizeof(float), hipMemcpyDeviceToHost, streams[i]));
    }
    // Destroy device streams
    for (int i = 0; i < n_streams; i++)
    {
        CHECK(hipStreamSynchronize(streams[i]));
        CHECK(hipStreamDestroy(streams[i]));
    }

    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());
    
    CHECK(hipMemcpy(out, d_out, size_out * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(d_in));
    CHECK(hipFree(d_out));
    CHECK(hipFree(d_weight));
    free(streams);
}

__host__ void cuda_manager::conv_forward_self1(const float *in, float *out, const float *w,
                                         const int B, const int C_in, const int C_out,
                                         const int H_in, const int W_in, const int K, const int n_streams,const int kernel)
{
  float *d_in;
  float *d_out;
  float *d_w;

  const int H_out = H_in - K + 1;
  const int W_out = W_in - K + 1;

  int inputArrayLength = B*C_in*H_in*W_in;
  int outputArrayLength = B*C_out*H_out*W_out;
  int kernelArrayLength = C_out*C_in*K*K;

  hipMalloc((void**) &d_in, inputArrayLength*sizeof(float));
  hipMalloc((void**) &d_w, kernelArrayLength*sizeof(float));
  hipMalloc((void**) &d_out, outputArrayLength*sizeof(float));

  hipMemcpy(d_in, in, inputArrayLength*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_w, w, kernelArrayLength*sizeof(float), hipMemcpyHostToDevice);

  int grid = ((H_out - 1) / TILE_WIDTH + 1) * ((W_out - 1) / TILE_WIDTH + 1);
  dim3 dimGrid(C_out, grid , B);
  dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);

  conv_forward_kernel<<<dimGrid, dimBlock>>>(d_in, d_out, d_w, C_in, C_out, H_in, W_in, K);

  CHECK(hipMemcpy(out, d_out, outputArrayLength * sizeof(float), hipMemcpyDeviceToHost));

  CHECK(hipFree(d_in));
  CHECK(hipFree(d_out));
  CHECK(hipFree(d_w));
}



